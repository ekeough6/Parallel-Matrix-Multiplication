#include <hip/hip_runtime.h>

#include <stdio.h>
#define BLOCK_SIZE 16
__global__ void gpu_matrix_multiply(float* a,float* b,float* c, int m, int n, int k)
  { 
  int row = blockIdx.y * blockDim.y + threadIdx.y; 
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float sum = 0;
  int i;
  if( col < k && row < m) 
  {
    for(i = 0; i < n; i++) 
    {
      sum += a[row * n + i] * b[i * k + col];
    }
    c[row * k + col] = sum;
  }
}

extern "C" void gpu_matrix_mult(float* a, float* b, float* c, int m, int n, int k) {
  float *d_a, *d_b, *d_c;
  hipMalloc((void **) &d_a, sizeof(float)*m*n);
  hipMalloc((void **) &d_b, sizeof(float)*n*k);
  hipMalloc((void **) &d_c, sizeof(float)*m*k);
  hipMemcpy(d_a, a, sizeof(float)*m*n, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float)*n*k, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, sizeof(float)*m*k, hipMemcpyHostToDevice);
  unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  gpu_matrix_multiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
  hipError_t i = hipMemcpy(c, d_c, sizeof(float)*m*k, hipMemcpyDeviceToHost);
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
