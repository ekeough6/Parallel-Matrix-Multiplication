#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
__global__ void gpu_matrix_multiply(float* a,float* b,float* c, int m, int n, int k)
  { 
  int row = blockIdx.y * blockDim.y + threadIdx.y; 
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int sum = 0;
  int i;
  if( col < k && row < m) 
  {
    for(i = 0; i < n; i++) 
    {
      sum += a[row * n + i] * b[i * k + col];
    }
    c[row * k + col] = sum;
  }
}

extern "C" void gpu_matrix_mult(float* a, float* b, float* c, int m, int n, int k) {
  float *d_a, *d_b, *d_c;
  hipMalloc((void **) &d_a, sizeof(float)*m*n);
  hipMalloc((void **) &d_b, sizeof(float)*n*k);
  hipMalloc((void **) &d_c, sizeof(float)*m*k);
  hipMemcpy(d_a, a, sizeof(float)*m*n, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(float)*n*k, hipMemcpyHostToDevice);
  unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  gpu_matrix_multiply<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
  hipMemcpy(c, d_c, sizeof(float)*m*k, hipMemcpyDeviceToHost);
}
